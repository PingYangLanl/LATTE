#include "hip/hip_runtime.h"
/*!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
! Copyright 2010.  Los Alamos National Security, LLC. This material was    !
! produced under U.S. Government contract DE-AC52-06NA25396 for Los Alamos !
! National Laboratory (LANL), which is operated by Los Alamos National     !
! Security, LLC for the U.S. Department of Energy. The U.S. Government has !
! rights to use, reproduce, and distribute this software.  NEITHER THE     !
! GOVERNMENT NOR LOS ALAMOS NATIONAL SECURITY, LLC MAKES ANY WARRANTY,     !
! EXPRESS OR IMPLIED, OR ASSUMES ANY LIABILITY FOR THE USE OF THIS         !
! SOFTWARE.  If software is modified to produce derivative works, such     !
! modified software should be clearly marked, so as not to confuse it      !
! with the version available from LANL.                                    !
!                                                                          !
! Additionally, this program is free software; you can redistribute it     !
! and/or modify it under the terms of the GNU General Public License as    !
! published by the Free Software Foundation; version 2.0 of the License.   !
! Accordingly, this program is distributed in the hope that it will be     !
! useful, but WITHOUT ANY WARRANTY; without even the implied warranty of   !
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU General !
! Public License for more details.                                         !
!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!*/

#include "Matrix.h"

REAL M_CGIterate(Matrix bo, Matrix p0, Matrix tmpmat, Matrix r0) {

  REAL error2=0.0;

  // Size is N
  int size = bo.DM;
  int blockCount = (int) ceil((float)size/(float)NUM_THREADS);
  int smemSize = NUM_THREADS * sizeof(REAL);
  REAL *device_error2;
  REAL *local_error2 = (REAL*)malloc(blockCount * sizeof(REAL));
    
  hipMalloc(&device_error2, blockCount * sizeof(REAL));

  hipSetDevice(0);

  CGIterateKernel<<<blockCount,NUM_THREADS,smemSize>>>(bo.DM, p0.Device[0], tmpmat.Device[0], r0.Device[0], bo.Device[0], device_error2);

  // Copy to local variable
  hipDeviceSynchronize();
  hipMemcpy(local_error2, device_error2, blockCount *sizeof(REAL), hipMemcpyDeviceToHost);
  hipFree(device_error2);

  for (int i = 0; i < blockCount; i++) {
    //printf ("i = %d local_error2 = %f\n", i, local_error2[i]);
    error2 += local_error2[i];
  }

  free(local_error2);

  return error2;
}

/*!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
! Copyright 2010.  Los Alamos National Security, LLC. This material was    !
! produced under U.S. Government contract DE-AC52-06NA25396 for Los Alamos !
! National Laboratory (LANL), which is operated by Los Alamos National     !
! Security, LLC for the U.S. Department of Energy. The U.S. Government has !
! rights to use, reproduce, and distribute this software.  NEITHER THE     !
! GOVERNMENT NOR LOS ALAMOS NATIONAL SECURITY, LLC MAKES ANY WARRANTY,     !
! EXPRESS OR IMPLIED, OR ASSUMES ANY LIABILITY FOR THE USE OF THIS         !
! SOFTWARE.  If software is modified to produce derivative works, such     !
! modified software should be clearly marked, so as not to confuse it      !
! with the version available from LANL.                                    !
!                                                                          !
! Additionally, this program is free software; you can redistribute it     !
! and/or modify it under the terms of the GNU General Public License as    !
! published by the Free Software Foundation; version 2.0 of the License.   !
! Accordingly, this program is distributed in the hope that it will be     !
! useful, but WITHOUT ANY WARRANTY; without even the implied warranty of   !
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU General !
! Public License for more details.                                         !
!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!*/

#include <stdlib.h>
#include <stdio.h>

#include "Matrix.h"

// CUBLAS handle - required
hipblasHandle_t *handle;
int maxdevices;
int ndevices;
int nblocks;
hipStream_t stream[MAX_GPUS];
hipEvent_t event[MAX_GPUS];

void M_Initialize( int NGPU ) {

  hipblasStatus_t status;

  // Get number of GPUs
  hipGetDeviceCount(&maxdevices);

  // Number of devices being used
  ndevices = NGPU;
//    fprintf(stdout, "Number of GPUs = %d \n", ndevices);

  // Number of blocks to spit up matrix into

  // Having the number of blocks = number of devices is usually a safe bet

    nblocks = 1*ndevices ;


  // Allow GPU 0 to access other GPUs
  if (ndevices > 1) {
    int accessible = 0;
    for (int d = 0; d < ndevices; d++) {
      for (int d2 = 0; d2 < ndevices; d2++) {
 //     printf("%d %d\n", d, d2);	
      
        if (d == d2) continue;

//	printf("2 %d %d\n", d, d2);	
        hipDeviceCanAccessPeer(&accessible, d, d2);
//	printf("2 %d %d %d\n", d, d2, accessible);	
//        if (accessible == 0) {
	  if (accessible) {
          hipSetDevice(d);
          hipDeviceEnablePeerAccess(d2, 0);
//          printf("GPU %d can access GPU %d\n", d, d2);
        }
      }
    }      
  }

    // Create and assign steams and events
  for (int d = 0; d < ndevices; ++d) {
    hipSetDevice(d);
    hipStreamCreate(&stream[d]);
    hipEventCreate(&event[d]);
  }
  hipSetDevice(0); 

  // Initialize per GPU with it's own handle
  // Associate steam with handle
  handle = (hipblasHandle_t*)malloc(ndevices * sizeof(hipblasHandle_t));
  for (int d = 0; d < ndevices; ++d) {
    hipSetDevice(d);

    status=hipblasCreate(&(handle[d]));
    if (status!=HIPBLAS_STATUS_SUCCESS) {
      if (status==HIPBLAS_STATUS_ALLOC_FAILED) {
        printf("Could not allocate resources for GPU %d!\n", d);
      }
      printf("CuBLAS init failedfor GPU %d!\n", d);
      exit(-1);
    }

    // Associate stream with cublas handle
    hipblasSetStream(handle[d], stream[d]);
  }

  // Create and assign steams and events
/*  for (int d = 0; d < ndevices; ++d) {
    hipSetDevice(d);
    hipStreamCreate(&stream[d]);
    hipEventCreate(&event[d]);
  } */
 
  hipSetDevice(0);

}


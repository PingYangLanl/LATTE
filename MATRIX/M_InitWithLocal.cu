/*!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
! Copyright 2010.  Los Alamos National Security, LLC. This material was    !
! produced under U.S. Government contract DE-AC52-06NA25396 for Los Alamos !
! National Laboratory (LANL), which is operated by Los Alamos National     !
! Security, LLC for the U.S. Department of Energy. The U.S. Government has !
! rights to use, reproduce, and distribute this software.  NEITHER THE     !
! GOVERNMENT NOR LOS ALAMOS NATIONAL SECURITY, LLC MAKES ANY WARRANTY,     !
! EXPRESS OR IMPLIED, OR ASSUMES ANY LIABILITY FOR THE USE OF THIS         !
! SOFTWARE.  If software is modified to produce derivative works, such     !
! modified software should be clearly marked, so as not to confuse it      !
! with the version available from LANL.                                    !
!                                                                          !
! Additionally, this program is free software; you can redistribute it     !
! and/or modify it under the terms of the GNU General Public License as    !
! published by the Free Software Foundation; version 2.0 of the License.   !
! Accordingly, this program is distributed in the hope that it will be     !
! useful, but WITHOUT ANY WARRANTY; without even the implied warranty of   !
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU General !
! Public License for more details.                                         !
!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!*/

#include "Matrix.h"

extern int ndevices;
extern int nblocks;

void M_InitWithLocal(Matrix &A, REAL *Local, int M, int N) {

  int Mpad, Npad;
  int cdev;
  int PadSize;
  // Get current device
  hipGetDevice(&cdev);

  A.M=M;
  A.N=N;

  PadSize = nblocks*16;

//  if ( nblocks <= 2 ) {	
    
    if ( sizeof(REAL) == 8) {
      
      // double precision case - multiples of 64 best
      
      if (N <= 736) {
	
	Npad = nblocks*((N-1)/nblocks + 1);
	
      } else if ( N > 736 ) {
	
	Npad = PadSize*((N-1)/PadSize + 1);
	
      } 
      
    }
    
    if (sizeof(REAL) == 4) {
      
      // Single precision dimensions 
      
      if (N <= 448) {
	
        Npad = nblocks*((N-1)/nblocks + 1);
	
      } else if ( N > 448 ) {
	
        Npad = PadSize*((N-1)/PadSize + 1);
	
      }
      
      
    }   
      
       

//  } else if ( nblocks > 2) {

    
//    Npad = nblocks*((N - 1)/nblocks + 1);

//    printf("%d %d %d\n", nblocks, N, Npad);

//  } 
  
//   Npad = N;
  
  A.DN = Npad;
  A.DM = A.DN*M/N;

//  printf("InitWithLocal: %d %d %d %d \n", M, Mpad, A.DM, A.DN);  

  
  A.Local=Local;

  for (int d = 0; d < ndevices; d++) {
    hipSetDevice(d);
    hipMalloc((void **)&A.Device[d], A.DM*A.DN*sizeof(REAL));
    hipMemset(A.Device[d], '\0', A.DM*A.DN*sizeof(REAL));
  }

  // Restore device
  hipSetDevice(cdev);
}

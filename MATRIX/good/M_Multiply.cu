#include "hip/hip_runtime.h"
/*!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
! Copyright 2010.  Los Alamos National Security, LLC. This material was    !
! produced under U.S. Government contract DE-AC52-06NA25396 for Los Alamos !
! National Laboratory (LANL), which is operated by Los Alamos National     !
! Security, LLC for the U.S. Department of Energy. The U.S. Government has !
! rights to use, reproduce, and distribute this software.  NEITHER THE     !
! GOVERNMENT NOR LOS ALAMOS NATIONAL SECURITY, LLC MAKES ANY WARRANTY,     !
! EXPRESS OR IMPLIED, OR ASSUMES ANY LIABILITY FOR THE USE OF THIS         !
! SOFTWARE.  If software is modified to produce derivative works, such     !
! modified software should be clearly marked, so as not to confuse it      !
! with the version available from LANL.                                    !
!                                                                          !
! Additionally, this program is free software; you can redistribute it     !
! and/or modify it under the terms of the GNU General Public License as    !
! published by the Free Software Foundation; version 2.0 of the License.   !
! Accordingly, this program is distributed in the hope that it will be     !
! useful, but WITHOUT ANY WARRANTY; without even the implied warranty of   !
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU General !
! Public License for more details.                                         !
!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!*/

#include "Matrix.h"

extern hipblasHandle_t* handle;
extern int ndevices;
extern int nblocks;
extern hipStream_t stream[];
extern hipEvent_t event[];

void M_Multiply(Matrix A, Matrix B, Matrix C) {

  hipSetDevice(0);

#if REALSIZE==4
  hipblasSgemm(handle[0], HIPBLAS_OP_N, HIPBLAS_OP_N, A.DM, B.DN, A.DN, &ONE, A.Device[0], A.DM, B.Device[0], B.DM, &ZERO, C.Device[0], C.DM);
#elif REALSIZE==8
  hipblasDgemm(handle[0], HIPBLAS_OP_N, HIPBLAS_OP_N, A.DM, B.DN, A.DN, &ONE, A.Device[0], A.DM, B.Device[0], B.DM, &ZERO, C.Device[0], C.DM);
#endif

}

void M_Multiply3(Matrix A, Matrix B, Matrix C) {

  hipSetDevice(0);

#if REALSIZE==4
  hipblasSgemm(handle[0], HIPBLAS_OP_N, HIPBLAS_OP_N, A.DM, B.DN, A.DN, &ONE, A.Device[0], A.DM, B.Device[0], B.DM, &ONE, C.Device[0], C.DM);
#elif REALSIZE==8
  hipblasDgemm(handle[0], HIPBLAS_OP_N, HIPBLAS_OP_N, A.DM, B.DN, A.DN, &ONE, A.Device[0], A.DM, B.Device[0], B.DM, &ONE, C.Device[0], C.DM);
#endif

}

void M_MultiplyTranspose(Matrix A, Matrix B, Matrix C) {

  hipSetDevice(0);

#if REALSIZE==4
  hipblasSgemm(handle[0], HIPBLAS_OP_T, HIPBLAS_OP_N, A.DM, B.DN, A.DN, &ONE, A.Device[0], A.DM, B.Device[0], B.DM, &ZERO, C.Device[0], C.DM);
#elif REALSIZE==8
  hipblasDgemm(handle[0], HIPBLAS_OP_T, HIPBLAS_OP_N, A.DM, B.DN, A.DN, &ONE, A.Device[0], A.DM, B.Device[0], B.DM, &ZERO, C.Device[0], C.DM);
#endif

}

void M_Multiply(REAL *scalar1, Matrix A, Matrix B, REAL *scalar2, Matrix C) {

  hipSetDevice(0);

#if REALSIZE==4
  hipblasSgemm(handle[0], HIPBLAS_OP_N, HIPBLAS_OP_N, A.DM, B.DN, A.DN, scalar1, A.Device[0], A.DM, B.Device[0], B.DM, scalar2, C.Device[0], C.DM);
#elif REALSIZE==8
  hipblasDgemm(handle[0], HIPBLAS_OP_N, HIPBLAS_OP_N, A.DM, B.DN, A.DN, scalar1, A.Device[0], A.DM, B.Device[0], B.DM, scalar2, C.Device[0], C.DM);
#endif

}

// Matrix multiplication on multiple GPUs
void M_MultiplyMgpu(REAL *scalar1, Matrix A, Matrix B, REAL *scalar2, Matrix C, Matrix C2) {

  int idevice = 0;          // GPU 0  
  int cdev;
  int ks;
  int iblock1, iblock2;     // indices for blocks to multiply    
  int oblock;               // index for output block
  int kblock, kpblock;      // which sub-block
  int sub = A.DN / nblocks; // size of each block

  // Save current device
  hipGetDevice(&cdev);

  //printf("DN = %d  nblocks = %d  sub = %d\n", A.DN, nblocks, sub);
  kblock = 0;
  ks = 0;

  for (int k = 0; k < nblocks; k++) {
    kpblock = 0;

    for (int kp = 0; kp < nblocks; kp++) {

      iblock1 = kblock * A.DN;
      iblock2 = kpblock * A.DN;

      oblock = kp * sub * sub + k * A.DN * sub;

      idevice = k % ndevices;
      hipSetDevice(idevice);

      //printf("idevice = %d  iblock1 = %d  iblock2 = %d  oblock = %d\n", idevice, iblock1, iblock2, oblock);

      // Associate stream with cublas call
      //hipblasSetStream(handle[idevice], stream[idevice]);

      // Multiply - results in sub x sub block
#if REALSIZE==4
      hipblasSgemm(handle[idevice], HIPBLAS_OP_N, HIPBLAS_OP_T, sub, sub, A.DN, scalar1, &A.Device[idevice]+iblock1, sub, B.Device[idevice]+iblock2, sub, scalar2, C2.Device[idevice]+oblock, sub);
#elif REALSIZE==8
      hipblasDgemm(handle[idevice], HIPBLAS_OP_N, HIPBLAS_OP_T, sub, sub, A.DN, scalar1, A.Device[idevice]+iblock1, sub, B.Device[idevice]+iblock2, sub, scalar2, C2.Device[idevice]+oblock, sub);
#endif

      ks++;
      kpblock += sub;
    }
    
    kblock += sub;
  }

  // Wait till all multiplies are done
  for (int d = 0; d < ndevices; ++d) {
    hipSetDevice(d);
    hipStreamSynchronize(stream[d]);
  }

  // Reassemble matrix blocks from stripes to grid
  M_AssembleMgpu(C, C2, sub);

/*
  // Sum up C's across GPUs
  hipSetDevice(0);
  if (ndevices > 1) {
    for (int d = 1; d < ndevices; ++d) {
      M_MultiplyScalarSumMgpu(d, &ONE, C2, C, stream);
    }
  } 

  // Wait till all sums are done
  hipStreamSynchronize(stream[0]);
*/

  // Restore device
  hipSetDevice(cdev);
}

void M_Multiply(REAL k, Matrix A, Matrix B) {

  int msize = A.DM * A.DN;
  int size = msize >> 1;
  int blockCount = (int) ceil((float)size/(float)NUM_THREADS);

  MultiplyScalarMatrixKernel<<<blockCount,NUM_THREADS>>>(msize, k, A.Device[0], B.Device[0]);

}

void M_MultiplyAdd(REAL k, Matrix A, REAL k2,  Matrix B, Matrix C) {

  int size = A.DM * A.DN;
  int blockCount = (int) ceil((float)size/(float)NUM_THREADS);

  MultiplyScalarMatrixAddKernel<<<blockCount,NUM_THREADS>>>(size, k, A.Device[0], k2, B.Device[0], C.Device[0]);
}

void M_MultiplyAdd(REAL k, Matrix A, Matrix B, Matrix C) {

  int size = A.DM * A.DN;
  int blockCount = (int) ceil((float)size/(float)NUM_THREADS);

  hipSetDevice(0);

  MultiplyScalarMatrixAddMatrixKernel<<<blockCount,NUM_THREADS>>>(size, k, A.Device[0], B.Device[0], C.Device[0]);
}

void M_MultiplySub(REAL k, Matrix A, REAL k2,  Matrix B, Matrix C) {

  int size = A.DM * A.DN;
  int blockCount = (int) ceil((float)size/(float)NUM_THREADS);

  hipSetDevice(0);

  MultiplyScalarMatrixSubKernel<<<blockCount,NUM_THREADS>>>(size, k, A.Device[0], k2, B.Device[0], C.Device[0]);
}

void M_MultiplySub(REAL k, Matrix A, Matrix B, Matrix C) {

  int size = A.DM * A.DN;
  int blockCount = (int) ceil((float)size/(float)NUM_THREADS);

  hipSetDevice(0);

  MultiplyScalarMatrixSubMatrixKernel<<<blockCount,NUM_THREADS>>>(size, k, A.Device[0], B.Device[0], C.Device[0]);
}


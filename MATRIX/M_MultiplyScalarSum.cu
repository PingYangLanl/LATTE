/*!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
! Copyright 2010.  Los Alamos National Security, LLC. This material was    !
! produced under U.S. Government contract DE-AC52-06NA25396 for Los Alamos !
! National Laboratory (LANL), which is operated by Los Alamos National     !
! Security, LLC for the U.S. Department of Energy. The U.S. Government has !
! rights to use, reproduce, and distribute this software.  NEITHER THE     !
! GOVERNMENT NOR LOS ALAMOS NATIONAL SECURITY, LLC MAKES ANY WARRANTY,     !
! EXPRESS OR IMPLIED, OR ASSUMES ANY LIABILITY FOR THE USE OF THIS         !
! SOFTWARE.  If software is modified to produce derivative works, such     !
! modified software should be clearly marked, so as not to confuse it      !
! with the version available from LANL.                                    !
!                                                                          !
! Additionally, this program is free software; you can redistribute it     !
! and/or modify it under the terms of the GNU General Public License as    !
! published by the Free Software Foundation; version 2.0 of the License.   !
! Accordingly, this program is distributed in the hope that it will be     !
! useful, but WITHOUT ANY WARRANTY; without even the implied warranty of   !
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU General !
! Public License for more details.                                         !
!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!*/

#include "Matrix.h"

extern hipblasHandle_t* handle;
extern int ndevices;
extern int nblocks;
extern hipStream_t stream[];
extern hipEvent_t event[];

void M_MultiplyScalarSum(REAL *scalar, Matrix A, Matrix B) {

  hipSetDevice(0);

#if REALSIZE==4
  hipblasSaxpy(handle[0], A.DM*A.DN, scalar, A.Device[0], 1, B.Device[0], 1);
#elif REALSIZE==8
  hipblasDaxpy(handle[0], A.DM*A.DN, scalar, A.Device[0], 1, B.Device[0], 1);
#endif

}

void M_MultiplyScalarSumMgpu(int idevice, REAL *scalar, Matrix A, Matrix B, hipStream_t stream[]) {

  hipSetDevice(0);

  hipMemcpy(A.Device[0], B.Device[idevice], A.DM*A.DN*sizeof(REAL), hipMemcpyDefault); 

#if REALSIZE==4
  hipblasSaxpy(handle[0], A.DM*A.DN, scalar, A.Device[0], 1, B.Device[0], 1);
#elif REALSIZE==8
  hipblasDaxpy(handle[0], A.DM*A.DN, scalar, A.Device[0], 1, B.Device[0], 1);
#endif

}

void M_MultiplyScalarSumMgpu(REAL *scalar, Matrix A, Matrix B) {

  int cdev;
  int sub = A.DN / nblocks;
  int size = A.DN * sub;

  // Get current device
  hipGetDevice(&cdev);

  for (int d = 0; d < ndevices; ++d) {

    hipSetDevice(d);

    for (int b = d; b < nblocks; b += ndevices) {

#if REALSIZE==4
      hipblasSaxpy(handle[d], size, scalar, A.Device[d]+b*size, 1, B.Device[d]+b*size, 1);
#elif REALSIZE==8
      hipblasDaxpy(handle[d], size, scalar, A.Device[d]+b*size, 1, B.Device[d]+b*size, 1);
#endif

    }
  }

  // Wait till done
  M_Wait();

  // Restore device
  hipSetDevice(cdev);
}

void M_MultiplyScalarSumMgpu(REAL *scalar, Matrix A, Matrix B, int d) {

  int sub = A.DN / nblocks;
  int size = A.DN * sub;
  

  for (int b = d; b < nblocks; b += ndevices) {

#if REALSIZE==4
    hipblasSaxpy(handle[d], size, scalar, A.Device[d]+b*size, 1, B.Device[d]+b*size, 1);
#elif REALSIZE==8
    hipblasDaxpy(handle[d], size, scalar, A.Device[d]+b*size, 1, B.Device[d]+b*size, 1);
#endif

    }
}

 
void M_MultiplyScalarSum(int i, REAL *scalar, Matrix A, Matrix B) {

  hipSetDevice(0);

#if REALSIZE==4
  hipblasSaxpy(handle[0], A.DM, scalar, &A.Device[0][i*A.DM], 1, &B.Device[0][i*B.DM], 1);
#elif REALSIZE==8
  hipblasDaxpy(handle[0], A.DM, scalar, &A.Device[0][i*A.DM], 1, &B.Device[0][i*B.DM], 1);
#endif

}

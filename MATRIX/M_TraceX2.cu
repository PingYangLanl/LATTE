#include "hip/hip_runtime.h"
/*!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
! Copyright 2010.  Los Alamos National Security, LLC. This material was    !
! produced under U.S. Government contract DE-AC52-06NA25396 for Los Alamos !
! National Laboratory (LANL), which is operated by Los Alamos National     !
! Security, LLC for the U.S. Department of Energy. The U.S. Government has !
! rights to use, reproduce, and distribute this software.  NEITHER THE     !
! GOVERNMENT NOR LOS ALAMOS NATIONAL SECURITY, LLC MAKES ANY WARRANTY,     !
! EXPRESS OR IMPLIED, OR ASSUMES ANY LIABILITY FOR THE USE OF THIS         !
! SOFTWARE.  If software is modified to produce derivative works, such     !
! modified software should be clearly marked, so as not to confuse it      !
! with the version available from LANL.                                    !
!                                                                          !
! Additionally, this program is free software; you can redistribute it     !
! and/or modify it under the terms of the GNU General Public License as    !
! published by the Free Software Foundation; version 2.0 of the License.   !
! Accordingly, this program is distributed in the hope that it will be     !
! useful, but WITHOUT ANY WARRANTY; without even the implied warranty of   !
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU General !
! Public License for more details.                                         !
!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!*/

#include "Matrix.h"

REAL M_TraceX2(Matrix A) {

  int size = (A.DM * A.DN) >> 1; // M*N/2
  int blockCount = (int) ceil((float)size/(float)NUM_THREADS);
  int smemSize = NUM_THREADS * sizeof(REAL);
  REAL *device_trace;
  REAL *local_trace = (REAL*)malloc(blockCount * sizeof(REAL));

  REAL trace=ZERO;

  hipMalloc(&device_trace, blockCount * sizeof(REAL));

  hipSetDevice(0);

  MatrixFastTraceX2Kernel<<<blockCount,NUM_THREADS,smemSize>>>(A.DM*A.DN, A.Device[0], device_trace);

  // Copy to local variable
  hipDeviceSynchronize();
  hipMemcpy(local_trace, device_trace, blockCount * sizeof(REAL), hipMemcpyDeviceToHost);
  hipFree(device_trace);

  for (int i = 0; i < blockCount; i++) {
    trace += local_trace[i];
  }

  free(local_trace);

  return trace;
}

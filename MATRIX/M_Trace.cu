#include "hip/hip_runtime.h"
/*!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
! Copyright 2010.  Los Alamos National Security, LLC. This material was    !
! produced under U.S. Government contract DE-AC52-06NA25396 for Los Alamos !
! National Laboratory (LANL), which is operated by Los Alamos National     !
! Security, LLC for the U.S. Department of Energy. The U.S. Government has !
! rights to use, reproduce, and distribute this software.  NEITHER THE     !
! GOVERNMENT NOR LOS ALAMOS NATIONAL SECURITY, LLC MAKES ANY WARRANTY,     !
! EXPRESS OR IMPLIED, OR ASSUMES ANY LIABILITY FOR THE USE OF THIS         !
! SOFTWARE.  If software is modified to produce derivative works, such     !
! modified software should be clearly marked, so as not to confuse it      !
! with the version available from LANL.                                    !
!                                                                          !
! Additionally, this program is free software; you can redistribute it     !
! and/or modify it under the terms of the GNU General Public License as    !
! published by the Free Software Foundation; version 2.0 of the License.   !
! Accordingly, this program is distributed in the hope that it will be     !
! useful, but WITHOUT ANY WARRANTY; without even the implied warranty of   !
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU General !
! Public License for more details.                                         !
!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!*/

#include "Matrix.h"

extern int ndevices;
extern int nblocks;
extern hipStream_t stream[];
extern hipEvent_t event[];

REAL M_Trace(Matrix A) {

  // Size is N/2
  int size = A.DM >> 1;
  int blockCount = (int) ceil((float)size/(float)NUM_THREADS);
  int smemSize = NUM_THREADS * sizeof(REAL);
  REAL *device_trace;
  REAL *local_trace = (REAL*)malloc(blockCount * sizeof(REAL));

  REAL trace=ZERO;

/*
  printf("N/2 = %d\n", size);
  printf("blockCount = %d\n", blockCount);
  printf("smemSize = %d\n ", smemSize);
  printf("NUM_THREADS = %d\n", NUM_THREADS);  
  printf("sizeof(REAL) = %ld\n", sizeof(REAL));
  printf("DM = %d\n", A.DM);
*/
 
  hipMalloc(&device_trace, blockCount * sizeof(REAL));

  hipSetDevice(0);

  MatrixFastTraceKernel<<<blockCount,NUM_THREADS,smemSize>>>(A.DM, A.DM, A.Device[0], device_trace, 0);

  // Copy to local variable
  hipDeviceSynchronize();
  hipMemcpy(local_trace, device_trace, blockCount * sizeof(REAL), hipMemcpyDeviceToHost);
  hipFree(device_trace);

  for (int i = 0; i < blockCount; i++) {
    trace += local_trace[i];
  }

  free(local_trace);

  return trace;
}

// This does not work correctly
REAL M_TraceMgpu(Matrix A, int idevice) {

  // Size is N/2
  int sub = A.DN / ndevices;
  int bsize = A.DN * sub;
  int size = sub >> 1;
  int blockCount = (int) ceil((float)size/(float)NUM_THREADS);
  int smemSize = NUM_THREADS * sizeof(REAL);
  REAL *device_trace;
  REAL *local_trace = (REAL*)malloc(blockCount * sizeof(REAL));

  REAL trace=ZERO;
  int cdev;

  // Get current device
  hipGetDevice(&cdev);

/*
  printf("N/2 = %d\n", size);
  printf("blockCount = %d\n", blockCount);
  printf("smemSize = %d\n ", smemSize);
  printf("NUM_THREADS = %d\n", NUM_THREADS);  
  printf("sizeof(REAL) = %ld\n", sizeof(REAL));
  printf("DM = %d\n", A.DM);
*/

  hipMalloc(&device_trace, blockCount * sizeof(REAL));

  hipSetDevice(idevice);

  MatrixFastTraceKernel<<<blockCount,NUM_THREADS,smemSize>>>(A.DM, sub, A.Device[idevice]+idevice*bsize, device_trace, idevice*sub);

  // Copy to local variable
  hipDeviceSynchronize();
  hipMemcpy(local_trace, device_trace, blockCount * sizeof(REAL), hipMemcpyDeviceToHost);
  hipFree(device_trace);

  for (int i = 0; i < blockCount; i++) {
    trace += local_trace[i];
  }

  free(local_trace);

  // Restore device
  hipSetDevice(cdev);

  return trace;
}

REAL M_TraceMgpu(Matrix A) {

  // Size is N * block size
  int sub = A.DN / nblocks;
  int bsize = A.DN * sub;
  int size = sub >> 1;
  int blockCount = (int) ceil((float)size/(float)NUM_THREADS);
  int smemSize = NUM_THREADS * sizeof(REAL);
  REAL *device_trace;
  REAL *local_trace = (REAL*)malloc(blockCount * sizeof(REAL));

  REAL trace=ZERO;
  int cdev;

  // Get current device
  hipGetDevice(&cdev);

/*
  printf("N/2 = %d\n", size);
  printf("blockCount = %d\n", blockCount);
  printf("smemSize = %d\n ", smemSize);
  printf("NUM_THREADS = %d\n", NUM_THREADS);
  printf("sizeof(REAL) = %ld\n", sizeof(REAL));
  printf("DM = %d\n", A.DM);
*/

  // For all GPUs
  for (int d = 0; d < ndevices; ++d) {

    hipSetDevice(d);

    hipMalloc(&device_trace, blockCount * sizeof(REAL));

    for (int b = d; b < nblocks; b+=ndevices) {

      MatrixFastTraceKernel<<<blockCount,NUM_THREADS,smemSize>>>(A.DM, sub, A.Device[d]+b*bsize, device_trace, b*sub);

      // Copy to local variable
      hipDeviceSynchronize();
      hipMemcpy(local_trace, device_trace, blockCount * sizeof(REAL), hipMemcpyDeviceToHost);

      for (int i = 0; i < blockCount; i++) {
        trace += local_trace[i];
      }
    }
    hipFree(device_trace);
  }

  free(local_trace);

  // Restore device
  hipSetDevice(cdev);

  return trace;
}

